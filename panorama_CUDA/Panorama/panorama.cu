#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES

#include <iostream>
#include <math.h>
#include <algorithm>
#include <string>
#include <chrono>
#include <thread>

#ifdef _WIN32
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#endif

// Include intel TBB if we're not running on a CUDA system (CUDA is currently Windows only, TBB Linux only)
#ifdef __linux__
#include <unistd.h>
#include <tbb/parallel_for.h>
#include <tbb/blocked_range.h>
#endif

#include "CImg.h"
using namespace cimg_library;

// Input parameters
int iflag, oflag, hflag, rflag, cflag;
char *ivalue, *ovalue;
int edge = 512;

#ifdef _WIN32
inline static void HandleError(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		//exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__device__
void OutImgToXYZ(int i, int j, int face, int edge, double *x, double *y, double *z) {
	auto a = 2.0 * i / edge;
	auto b = 2.0 * j / edge;

	if (face == 0) { // back
		*x = -1;
		*y = 1 - a;
		*z = 3 - b;
	}
	else if (face == 1) { // left
		*x = a - 3;
		*y = -1;
		*z = 3 - b;
	}
	else if (face == 2) { // front
		*x = 1;
		*y = a - 5;
		*z = 3 - b;
	}
	else if (face == 3) { // right
		*x = 7 - a;
		*y = 1;
		*z = 3 - b;
	}
	else if (face == 4) { // top
		*x = b - 1;
		*y = a - 5;
		*z = 1;
	}
	else if (face == 5) { // bottom
		*x = 5 - b;
		*y = a - 5;
		*z = -1;
	}
}

__global__
void ConvertBack(unsigned char *imgIn, unsigned char *front, unsigned char *back,
	unsigned char *left, unsigned char *right, unsigned char *top, unsigned char *bottom,
	int width, int height, int rvalue)
{
	long TstripWidth = rvalue * 4; // Use long in case we're using gigantic 32k+ images
	int edge = rvalue;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = (blockDim.x * gridDim.x);

	int i = index;
	int maxIndex = 0;
	for (; i < TstripWidth; i += stride) {
		int face = int(i / edge);
		int start = edge;
		int end = 2 * edge;

		if (i >= 2 * edge && i < 3 * edge) {
			start = 0;
			end = 3 * edge;
		}

		for (int j = start; j < end; j++) {
			if (j < edge) {
				face = 4;
			}
			else if (j > 2 * edge) {
				face = 5;
			}
			else {
				face = int(i / edge);
			}
			double x, y, z;
			OutImgToXYZ(i, j, face, edge, &x, &y, &z);

			unsigned char *facePtr;
			switch (face) {
			case 0:
				facePtr = back;
				break;
			case 1:
				facePtr = left;
				break;
			case 2:
				facePtr = front;
				break;
			case 3:
				facePtr = right;
				break;
			case 4:
				facePtr = top;
				break;
			case 5:
				facePtr = bottom;
				break;
			}

			double theta = atan2(y, x);
			double r = hypot(x, y);
			double phi = atan2(z, r);
			double uf = (theta + M_PI) / M_PI * height;
			double vf = (M_PI_2 - phi) / M_PI * height;
			int ui = min(static_cast<int>(std::floor(uf)), width);
			int vi = min(static_cast<int>(std::floor(vf)), height);
			int u2 = min(ui + 1, width);
			int v2 = min(vi + 1, height);
			double mu = uf - ui, nu = vf - vi;
			mu = nu = 0;

			// This is the old "read" and "mix" operations unraveled
			// Take first R from ui/vi then mix with second R from u2/vi
			// Repeat for G/B and RGB again for ui/v2 u2/v2
			unsigned char Ra = unsigned char(imgIn[ui + vi * width + 0 * width*height] + (imgIn[u2 + vi * width + 0 * width*height] - imgIn[ui + vi * width + 0 * width*height]) * mu);
			unsigned char Ga = unsigned char(imgIn[ui + vi * width + 1 * width*height] + (imgIn[u2 + vi * width + 1 * width*height] - imgIn[ui + vi * width + 1 * width*height]) * mu);
			unsigned char Ba = unsigned char(imgIn[ui + vi * width + 2 * width*height] + (imgIn[u2 + vi * width + 2 * width*height] - imgIn[ui + vi * width + 2 * width*height]) * mu);
			unsigned char Rb = unsigned char(imgIn[ui + v2 * width + 0 * width*height] + (imgIn[u2 + v2 * width + 0 * width*height] - imgIn[ui + v2 * width + 0 * width*height]) * mu);
			unsigned char Gb = unsigned char(imgIn[ui + v2 * width + 1 * width*height] + (imgIn[u2 + v2 * width + 1 * width*height] - imgIn[ui + v2 * width + 1 * width*height]) * mu);
			unsigned char Bb = unsigned char(imgIn[ui + v2 * width + 2 * width*height] + (imgIn[u2 + v2 * width + 2 * width*height] - imgIn[ui + v2 * width + 2 * width*height]) * mu);
			// Finally mix Ra/Rb etc together for finally interpolated color
			unsigned char R = Ra + (Rb - Ra) * nu;
			unsigned char G = Ga + (Gb - Ga) * nu;
			unsigned char B = Ba + (Bb - Ba) * nu;

			// Based on T-Strip coordinates, mod to edge size and insert into appropriate face
			int idx = ((i%edge) + (j%edge)*edge);
			// CImg uses planar RGBA storage, hence n*edge*edge for each value
			facePtr[idx + 0 * edge*edge] = R;
			facePtr[idx + 1 * edge*edge] = G;
			facePtr[idx + 2 * edge*edge] = B;
			facePtr[idx + 3 * edge*edge] = 255;
		}
	}
}

__device__
void GetFaceStartEnd(int face, int srcWidth, int srcHeight, int edgeSize, 
	int *xStart, int *xEnd, int *yStart, int *yEnd)
{
	// back, left, front, right, top, bottom
	switch (face) {
	case 0: // back
		*xStart = 0;
		*xEnd = edgeSize;
		*yStart = edgeSize;
		*yEnd = edgeSize * 2;
		break;
	case 1: // left
		*xStart = edgeSize;
		*xEnd = edgeSize*2;
		*yStart = edgeSize;
		*yEnd = edgeSize * 2;
		break;
	case 2:
		*xStart = edgeSize * 2;
		*xEnd = edgeSize * 3;
		*yStart = edgeSize;
		*yEnd = edgeSize * 2;
		break;
	case 3:
		*xStart = edgeSize * 3;
		*xEnd = edgeSize * 4;
		*yStart = edgeSize;
		*yEnd = edgeSize * 2;
		break;
	case 4:
		*xStart = edgeSize * 2;
		*xEnd = edgeSize * 3;
		*yStart = 0;
		*yEnd = edgeSize;
		break;
	case 5:
		*xStart = edgeSize * 2;
		*xEnd = edgeSize * 3;
		*yStart = edgeSize * 2;
		*yEnd = edgeSize * 3;
		break;

	}
}

__global__
void ConvertFace(unsigned char *imgIn, unsigned char *imgOut, int face, int width, int height, int rvalue)
{
	long TstripWidth = rvalue * 4; // Use long in case we're using gigantic 32k+ images
	int edge = rvalue;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = (blockDim.x * gridDim.x);
	int xStart, xEnd, yStart, yEnd;
	GetFaceStartEnd(face, width, height, edge, &xStart, &xEnd, &yStart, &yEnd);

	int i = index + xStart;
	int maxIndex = 0;
	for (; i < xEnd; i += stride) {
		for (int j = yStart; j < yEnd; j++) {
			
			double x, y, z;
			OutImgToXYZ(i, j, face, edge, &x, &y, &z);

			double theta = atan2(y, x);
			double r = hypot(x, y);
			double phi = atan2(z, r);
			double uf = (theta + M_PI) / M_PI * height;
			double vf = (M_PI_2 - phi) / M_PI * height;
			int ui = min(static_cast<int>(std::floor(uf)), width);
			int vi = min(static_cast<int>(std::floor(vf)), height);
			int u2 = min(ui + 1, width);
			int v2 = min(vi + 1, height);
			double mu = uf - ui, nu = vf - vi;
			mu = nu = 0;

			// This is the old "read" and "mix" operations unraveled
			// Take first R from ui/vi then mix with second R from u2/vi
			// Repeat for G/B and RGB again for ui/v2 u2/v2
			unsigned char Ra = unsigned char(imgIn[ui + vi * width + 0 * width*height] + (imgIn[u2 + vi * width + 0 * width*height] - imgIn[ui + vi * width + 0 * width*height]) * mu);
			unsigned char Ga = unsigned char(imgIn[ui + vi * width + 1 * width*height] + (imgIn[u2 + vi * width + 1 * width*height] - imgIn[ui + vi * width + 1 * width*height]) * mu);
			unsigned char Ba = unsigned char(imgIn[ui + vi * width + 2 * width*height] + (imgIn[u2 + vi * width + 2 * width*height] - imgIn[ui + vi * width + 2 * width*height]) * mu);
			unsigned char Rb = unsigned char(imgIn[ui + v2 * width + 0 * width*height] + (imgIn[u2 + v2 * width + 0 * width*height] - imgIn[ui + v2 * width + 0 * width*height]) * mu);
			unsigned char Gb = unsigned char(imgIn[ui + v2 * width + 1 * width*height] + (imgIn[u2 + v2 * width + 1 * width*height] - imgIn[ui + v2 * width + 1 * width*height]) * mu);
			unsigned char Bb = unsigned char(imgIn[ui + v2 * width + 2 * width*height] + (imgIn[u2 + v2 * width + 2 * width*height] - imgIn[ui + v2 * width + 2 * width*height]) * mu);
			// Finally mix Ra/Rb etc together for finally interpolated color
			unsigned char R = Ra + (Rb - Ra) * nu;
			unsigned char G = Ga + (Gb - Ga) * nu;
			unsigned char B = Ba + (Bb - Ba) * nu;

			// Based on T-Strip coordinates, mod to edge size and insert into appropriate face
			int idx = ((i%edge) + (j%edge)*edge);
			// CImg uses planar RGBA storage, hence n*edge*edge for each value
			imgOut[idx + 0 * edge*edge] = R;
			imgOut[idx + 1 * edge*edge] = G;
			imgOut[idx + 2 * edge*edge] = B;
			imgOut[idx + 3 * edge*edge] = 255;
		}
	}
}
#endif

void ImgToXYZ(int i, int j, int face, int edge, double *x, double *y, double *z) {
	auto a = 2.0 * i / edge;
	auto b = 2.0 * j / edge;

	if (face == 0) { // back
		*x = -1;
		*y = 1 - a;
		*z = 3 - b;
	}
	else if (face == 1) { // left
		*x = a - 3;
		*y = -1;
		*z = 3 - b;
	}
	else if (face == 2) { // front
		*x = 1;
		*y = a - 5;
		*z = 3 - b;
	}
	else if (face == 3) { // right
		*x = 7 - a;
		*y = 1;
		*z = 3 - b;
	}
	else if (face == 4) { // top
		*x = b - 1;
		*y = a - 5;
		*z = 1;
	}
	else if (face == 5) { // bottom
		*x = 5 - b;
		*y = a - 5;
		*z = -1;
	}
}

/**
**	Convert panorama using an inverse pixel transformation on CPU
**/
void ConvertCPU(unsigned char *imgIn, unsigned char **imgOut, int width, int height) {
	int TotalWidth = edge * 4; // Total width of the T-Strip image
	int start = 0;
	int end = 0;

							   // i/j are T-Strip coordinates, *not* source image coordinates
#ifdef _WIN32
							   // Pardon the sloppy indent on these two loops, it just makes it easier to read with the macros
	for (int i = 0; i < TotalWidth; i++) {
#elif __linux__
	tbb::parallel_for(blocked_range<size_t>(0, TotalWidth, 1), [&](const blocked_range<size_t>& range) {
		for (size_t i = range.begin(); i < range.end(); i++) {
#endif
			int face = int(i / edge);
			start = (i >= 2 * edge && i < 3 * edge) ? 0 : edge;
			end = (i >= 2 * edge && i < 3 * edge) ? edge * 3 : edge * 2;

			// Range start/end determine where in the T-strip to look vertically
			for (int j = start; j < end; ++j) {
				if (j < edge) { // Check if we're above the middle of the strip, then it's the top face
					face = 4;
				}
				else if (j > 2 * edge) { // If we're below the middle of the strip, bottom face
					face = 5;
				}
				else {
					face = int(i / edge); // In the middle of the strip, determine by ratio what face we have
				}

				// Covert T-Strip coordinates to unit-cube coordinates
				double x, y, z;
				ImgToXYZ(i, j, face, edge, &x, &y, &z);

				// Convert unit-cube coordinates into projected unit-sphere coordinates
				double theta = std::atan2(y, x);
				double r = std::hypot(x, y);
				double phi = std::atan2(z, r);
				double uf = (theta + M_PI) / M_PI * height;
				double vf = (M_PI_2 - phi) / M_PI * height;
				int ui = std::min(static_cast<int>(std::floor(uf)), width);
				int vi = std::min(static_cast<int>(std::floor(vf)), height);
				int u2 = std::min(ui + 1, width);
				int v2 = std::min(vi + 1, height);
				double mu = uf - ui, nu = vf - vi;
				mu = nu = 0;

				// This is the old "read" and "mix" operations unraveled
				// Take first R from ui/vi then mix with second R from u2/vi
				// Repeat for G/B and RGB again for ui/v2 u2/v2
				unsigned char Ra = unsigned char(imgIn[ui + vi * width + 0 * width*height] + (imgIn[u2 + vi * width + 0 * width*height] - imgIn[ui + vi * width + 0 * width*height]) * mu);
				unsigned char Ga = unsigned char(imgIn[ui + vi * width + 1 * width*height] + (imgIn[u2 + vi * width + 1 * width*height] - imgIn[ui + vi * width + 1 * width*height]) * mu);
				unsigned char Ba = unsigned char(imgIn[ui + vi * width + 2 * width*height] + (imgIn[u2 + vi * width + 2 * width*height] - imgIn[ui + vi * width + 2 * width*height]) * mu);
				unsigned char Rb = unsigned char(imgIn[ui + v2 * width + 0 * width*height] + (imgIn[u2 + v2 * width + 0 * width*height] - imgIn[ui + v2 * width + 0 * width*height]) * mu);
				unsigned char Gb = unsigned char(imgIn[ui + v2 * width + 1 * width*height] + (imgIn[u2 + v2 * width + 1 * width*height] - imgIn[ui + v2 * width + 1 * width*height]) * mu);
				unsigned char Bb = unsigned char(imgIn[ui + v2 * width + 2 * width*height] + (imgIn[u2 + v2 * width + 2 * width*height] - imgIn[ui + v2 * width + 2 * width*height]) * mu);
				// Finally mix Ra/Rb etc together for finally interpolated color
				unsigned char R = Ra + (Rb - Ra) * nu;
				unsigned char G = Ga + (Gb - Ga) * nu;
				unsigned char B = Ba + (Bb - Ba) * nu;

				// Based on T-Strip coordinates, mod to edge size and insert into appropriate face
				int idx = ((i%edge) + (j%edge)*edge);
				// CImg uses planar RGBA storage, hence n*edge*edge for each value
				imgOut[face][idx + 0 * edge*edge] = R;
				imgOut[face][idx + 1 * edge*edge] = G;
				imgOut[face][idx + 2 * edge*edge] = B;
				imgOut[face][idx + 3 * edge*edge] = 255;
			}
		}
#ifdef __linux__
	});
#endif
}


int parseParameters(int argc, char *argv[]) {
	iflag = oflag = hflag = rflag = cflag = 0;
	ivalue = ovalue = NULL;
	int c;

#ifdef _WIN32
	for (int i = 1; i < argc; i++) {
		if (argv[i] == std::string("-i")) {
			iflag = 1;
			ivalue = argv[++i];
		}
		if (argv[i] == std::string("-o")) {
			oflag = 1;
			ovalue = argv[++i];
		}
		if (argv[i] == std::string("-r")) {
			rflag = 1;
			edge = std::stoi(argv[++i]);
		}
		if (argv[i] == std::string("-c")) {
			cflag = 1;
		}
		if (argv[i] == std::string("-h")) {
			fprintf(stderr, "Usage:\n\t -i <input file>\n\t -o <output file(s)>\n\t -r <edge size>\n\t -c (enable CUDA)\n");
			abort();
		}
	}
#elif __linux__
	opterr = 0;
	while ((c = getopt(argc, argv, "i:o:r:")) != -1)
		switch (c) {
		case 'i':
			// input file
			iflag = 1;
			ivalue = optarg;
			break;
		case 'o':
			oflag = 1;
			ovalue = optarg;
			break;
		case 'r':
			rflag = 1;
			edge = std::stoi(optarg);
			break;
		case '?':
			if (optopt == 'i' || optopt == 'o' || optopt == 'r')
				fprintf(stderr, "Option -%c requires an argument.\n", optopt);
			else if (isprint(optopt))
				fprintf(stderr, "Unknown option `-%c'.\n", optopt);
			else
				fprintf(stderr, "Unknown option character `\\x%x'.\n", optopt);
			return 1;
		default:
			abort();
		}
#endif

	if (iflag == 0 || oflag == 0) {
		fprintf(stderr, "No inputs or outputs specified: %d / %d\n", iflag, oflag);
		abort();
		return 1;
	}
	return 0;
}


int main(int argc, char *argv[])
{

	parseParameters(argc, argv);

	CImg<unsigned char> ImgIn(ivalue);

	// Create output images
	CImg<unsigned char>* CImgOut[6];
	unsigned char* imgOut[6];
	for (int i = 0; i < 6; ++i) {
		CImgOut[i] = new CImg<unsigned char>(edge, edge, 1, 4, 255);
		imgOut[i] = (unsigned char*)CImgOut[i]->data();
	}
	std::chrono::high_resolution_clock::time_point total = std::chrono::high_resolution_clock::now();
	if (cflag) {
		int InSize = ImgIn.size();
		int width = ImgIn.width();
		int height = ImgIn.height();
		unsigned char *d_ImgIn, *d_ImgFront, *d_ImgBack, *d_ImgLeft, *d_ImgRight, *d_ImgTop, *d_ImgBottom;
		HANDLE_ERROR(hipMallocManaged((void**)&d_ImgIn, InSize * sizeof(unsigned char)));
		std::memcpy(d_ImgIn, ImgIn.data(), InSize * sizeof(unsigned char));
		ImgIn.clear();

		size_t total_memory, free_memory;
		HANDLE_ERROR(hipMemGetInfo(&free_memory, &total_memory) );
		printf("Total Memory: %lld\n Free Memory: %lld\n", total_memory, free_memory);

		long outSize = edge * edge * 4;
		printf("Insize memory: %d\n Outsize memory each: %lld\n Outsize memory total: %lld\n", InSize, outSize, outSize * 6);

		if (outSize > free_memory) {
			fprintf(stderr, "Not enough memory free on GPU device, please run without -c flag to compute on CPU");
			return -1;
		}

		if (outSize * 6 > free_memory) {
			//std::thread threads[6];
			fprintf(stderr, "Not enough memory on device to run all faces in parallel, running one face at a time\n");
			unsigned char *d_ImgOut, *h_ImgOut;
			//HANDLE_ERROR(hipMallocManaged((void**)&d_ImgOut, outSize * sizeof(unsigned char)));
			HANDLE_ERROR(hipMalloc((void**)&d_ImgOut, outSize * sizeof(unsigned char)));
			HANDLE_ERROR(hipHostMalloc((void**)&h_ImgOut, outSize * sizeof(unsigned char)));
			std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
			int blockSize = 256;
			int blockCount = (edge + blockSize - 1) / blockSize;
			printf("Total threads: %d\n", blockSize * blockCount);
			for (int i = 0; i < 6; i++) {
				fprintf(stderr, "Starting face %d...\n", i);
				ConvertFace <<<blockCount, blockSize >>> (d_ImgIn, d_ImgOut, i, width, height, edge);
				HANDLE_ERROR(hipDeviceSynchronize());
				HANDLE_ERROR(hipDeviceSynchronize());
				//std::memcpy(imgOut[i], d_ImgOut, outSize * sizeof(unsigned char));
				HANDLE_ERROR(hipMemcpy(h_ImgOut, d_ImgOut, outSize * sizeof(unsigned char), hipMemcpyDeviceToHost));
				std::memcpy(imgOut[i], h_ImgOut, outSize * sizeof(unsigned char));
				std::string fname = std::string(ovalue) + "_" + std::to_string(i) + ".png";
				CImgOut[i]->save_png(fname.c_str());
				CImgOut[i]->clear();
				//threads[i] = std::thread([&]() {
				//	std::string fname = std::string(ovalue) + "_" + std::to_string(i) + ".png";
				//	CImgOut[i]->save_png(fname.c_str());
				//	CImgOut[i]->clear();
				//});
			}
			HANDLE_ERROR(hipFree(d_ImgIn));
			HANDLE_ERROR(hipFree(d_ImgOut));
			HANDLE_ERROR(hipHostFree(h_ImgOut));
			fprintf(stderr, "Time to convert: %lldms\n", std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - t1).count());
			//for (int i = 0; i < 6; i++) {
			//	threads[i].join();
			//}
			//for (int i = 0; i < 6; i++) {
			//	std::string fname = std::string(ovalue) + "_" + std::to_string(i) + ".png";
			//	CImgOut[i]->save_png(fname.c_str());
			//}
			fprintf(stderr, "Total Time To Convert And Write: %lldms\n", std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - total).count());
		}
		else {

			HANDLE_ERROR(hipMallocManaged((void**)&d_ImgFront, outSize * sizeof(unsigned char)));
			HANDLE_ERROR(hipMemGetInfo(&free_memory, &total_memory));
			printf("Free Memory: %lld\n", free_memory);

			HANDLE_ERROR(hipMallocManaged((void**)&d_ImgBack, outSize * sizeof(unsigned char)));
			HANDLE_ERROR(hipMemGetInfo(&free_memory, &total_memory));
			printf("Free Memory: %lld\n", free_memory);

			HANDLE_ERROR(hipMallocManaged((void**)&d_ImgLeft, outSize * sizeof(unsigned char)));
			HANDLE_ERROR(hipMemGetInfo(&free_memory, &total_memory));
			printf("Free Memory: %lld\n", free_memory);

			HANDLE_ERROR(hipMallocManaged((void**)&d_ImgRight, outSize * sizeof(unsigned char)));
			HANDLE_ERROR(hipMemGetInfo(&free_memory, &total_memory));
			printf("Free Memory: %lld\n", free_memory);

			HANDLE_ERROR(hipMallocManaged((void**)&d_ImgTop, outSize * sizeof(unsigned char)));
			HANDLE_ERROR(hipMemGetInfo(&free_memory, &total_memory));
			printf("Free Memory: %lld\n", free_memory);

			HANDLE_ERROR(hipMallocManaged((void**)&d_ImgBottom, outSize * sizeof(unsigned char)));
			HANDLE_ERROR(hipMemGetInfo(&free_memory, &total_memory));
			printf("Free Memory: %lld\n", free_memory);

			std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
			int blockSize = 256;
			int blockCount = (ImgIn.width() + blockSize - 1) / blockSize;
			fprintf(stderr, "Starting...\n");
			ConvertBack << <blockCount, blockSize >> > (d_ImgIn, d_ImgFront, d_ImgBack, d_ImgLeft, d_ImgRight, d_ImgTop, d_ImgBottom, ImgIn.width(), ImgIn.height(), edge);
			HANDLE_ERROR(hipDeviceSynchronize());
			HANDLE_ERROR(hipFree(d_ImgIn));
			fprintf(stderr, "Time to convert: %lldms\n", std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - t1).count());
			fprintf(stderr, "Synchronized\n");


			// TODO: Loop this
			std::string filename = std::string(ovalue); // This is so stupid
			std::string copy = filename;
			copy.append("_0.png");
			printf("Writing Back...\n");
			std::memcpy(imgOut[0], d_ImgBack, outSize * sizeof(unsigned char));
			HANDLE_ERROR(hipFree(d_ImgBack));
			CImgOut[0]->save_png(copy.c_str());

			printf("Writing Front...\n");
			copy = filename;
			copy.append("_1.png");
			std::memcpy(imgOut[1], d_ImgFront, outSize * sizeof(unsigned char));
			HANDLE_ERROR(hipFree(d_ImgFront));
			CImgOut[1]->save_png(copy.c_str());

			printf("Writing Left...\n");
			copy = filename;
			copy.append("_2.png");
			std::memcpy(imgOut[2], d_ImgLeft, outSize * sizeof(unsigned char));
			HANDLE_ERROR(hipFree(d_ImgLeft));
			CImgOut[2]->save_png(copy.c_str());

			printf("Writing Right...\n");
			copy = filename;
			copy.append("_3.png");
			std::memcpy(imgOut[3], d_ImgRight, outSize * sizeof(unsigned char));
			HANDLE_ERROR(hipFree(d_ImgRight));
			CImgOut[3]->save_png(copy.c_str());

			printf("Writing Top...\n");
			copy = filename;
			copy.append("_4.png");
			std::memcpy(imgOut[4], d_ImgTop, outSize * sizeof(unsigned char));
			HANDLE_ERROR(hipFree(d_ImgTop));
			CImgOut[4]->save_png(copy.c_str());

			printf("Writing Bottom...\n");
			copy = filename;
			copy.append("_5.png");
			std::memcpy(imgOut[5], d_ImgBottom, outSize * sizeof(unsigned char));
			HANDLE_ERROR(hipFree(d_ImgBottom));
			CImgOut[5]->save_png(copy.c_str());
			fprintf(stderr, "Total Time To Convert And Write: %lldms\n", std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - total).count());
		}
	}
	else {
		ConvertCPU(ImgIn.data(), imgOut, ImgIn.width(), ImgIn.height());

		for (int i = 0; i < 6; i++) {
			std::string fname = std::string(ovalue) + "_" + std::to_string(i) + ".png";
			CImgOut[i]->save_png(fname.c_str());
		}
		fprintf(stderr, "Total Time To Convert And Write: %lldms\n", std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - total).count());
	}

	return 0;
}



