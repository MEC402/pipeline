#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#define GLM_ENABLE_EXPERIMENTAL

#include <iostream>
#include <math.h>
#include <algorithm>
#include <string>
#include <chrono>
#include <thread>
#include <glm/glm.hpp>
#include <glm/gtx/transform.hpp>
#include <glm/gtx/rotate_vector.hpp>

#ifdef _WIN32
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "png.h"
#endif

#ifdef __linux__
#include <unistd.h>
#include <tbb/parallel_for.h>
#include <tbb/blocked_range.h>
#endif

// If on windows: Make sure libpng nuget package is installed to make use of native png saving
// If on linux: Make sure libpng is linked against when compiling
#define cimg_use_png
#include "CImg.h"
using namespace cimg_library;

// Input parameters
int iflag, oflag, hflag, rflag, tflag, cflag;
char *ivalue, *ovalue;
int edge = 512;
double yaw = 0.0, pitch = 0.0, roll = 0.0;

#ifdef _WIN32
#pragma region CUDACALLS
inline static void HandleError(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		//exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__device__
void OutImgToXYZ(int i, int j, int face, int edge, double *x, double *y, double *z) {
	auto a = 2.0 * i / edge;
	auto b = 2.0 * j / edge;

	if (face == 0) { // back
		*x = -1;
		*y = 1 - a;
		*z = 3 - b;
	}
	else if (face == 1) { // left
		*x = a - 3;
		*y = -1;
		*z = 3 - b;
	}
	else if (face == 2) { // front
		*x = 1;
		*y = a - 5;
		*z = 3 - b;
	}
	else if (face == 3) { // right
		*x = 7 - a;
		*y = 1;
		*z = 3 - b;
	}
	else if (face == 4) { // top
		*x = b - 1;
		*y = a - 5;
		*z = 1;
	}
	else if (face == 5) { // bottom
		*x = 5 - b;
		*y = a - 5;
		*z = -1;
	}
}

// Overload for nesting Interpolate calls
__device__
inline unsigned char LinearInterpolate(float weight, unsigned char v1, unsigned char v2)
{
	return unsigned char(weight*v2 + (1.0f - weight) * v1);
}

// Expectes 1 weight and 2 values
__device__
inline unsigned char LinearInterpolate(float weight, unsigned char *values)
{
	return unsigned char(weight * (values[1]) + (1.0f - weight)*values[0]);
}

// Expects 2 weights and 4 values
__device__
inline unsigned char BilinearInterpolate(float *weight, unsigned char *values)
{
	unsigned char prime[2] = {
		LinearInterpolate(weight[1], &values[0]),
		LinearInterpolate(weight[1], &values[2])
	};
	return LinearInterpolate(weight[0], prime);
}

// Expects 3 weights and 8 values
__device__
inline unsigned char TrilinearInterpolate(float *weight, unsigned char *values)
{
	unsigned char prime[2] = {
		BilinearInterpolate(&(weight[0]), &(values[0])),
		BilinearInterpolate(&(weight[1]), &(values[4]))
	};
	return LinearInterpolate(weight[2], prime);
}

// Convert the entire cubemap at once
__global__
void ConvertBack(unsigned char *imgIn, unsigned char **imgOut, int width, int height, int rvalue)
{
	long TstripWidth = rvalue * 4; // Use long in case we're using gigantic 32k+ images
	int edge = rvalue;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = (blockDim.x * gridDim.x);

	int i = index;
	int maxIndex = 0;
	for (; i < TstripWidth; i += stride) {
		int face = int(i / edge);
		int start = edge;
		int end = 2 * edge;

		if (i >= 2 * edge && i < 3 * edge) {
			start = 0;
			end = 3 * edge;
		}

		for (int j = start; j < end; j++) {
			if (j < edge) {
				face = 4;
			}
			else if (j > 2 * edge) {
				face = 5;
			}
			else {
				face = int(i / edge);
			}
			double x, y, z;
			OutImgToXYZ(i, j, face, edge, &x, &y, &z);

			// This was originally a seperate function, but CUDA likes to do things in-place if it can.
			// Chaining device function calls leads to memory complexity and debugging CUDA is a real PITA
			double theta = atan2(y, x);
			double r = hypot(x, y);
			double phi = atan2(z, r);
			double uf = (theta + M_PI) / M_PI * height;
			double vf = (M_PI_2 - phi) / M_PI * height;
			int ui = min(static_cast<int>(std::floor(uf)), width);
			int vi = min(static_cast<int>(std::floor(vf)), height);
			int u2 = min(ui + 1, width);
			int v2 = min(vi + 1, height);
			int u3 = min(ui + 2, width);
			int v3 = min(vi + 2, height);
			int u4 = max(ui - 1, 0);
			int v4 = max(vi - 1, 0);
			int u[4] = { ui, u2, u3, u4 };
			int v[4] = { vi, v2, v3, v4 };

			unsigned char Rval[16];
			unsigned char Gval[16];
			unsigned char Bval[16];

			// Get the RGB values of our 16 pixels
			for (int a = 0; a < 4; a++) {
				for (int b = 0; b < 4; b++) {
					Rval[a * 4 + b] = imgIn[u[a] + v[b] * width + 0 * width*height];
					Gval[a * 4 + b] = imgIn[u[a] + v[b] * width + 1 * width*height];
					Bval[a * 4 + b] = imgIn[u[a] + v[b] * width + 2 * width*height];
				}
			}

			// Interpolate to get our desired single pixel color value
			float weight[3] = { 0.5f, 0.5f, 0.5f };
			unsigned char R = LinearInterpolate(weight[0], TrilinearInterpolate(weight, &Rval[0]), TrilinearInterpolate(weight, &Rval[8]));
			unsigned char G = LinearInterpolate(weight[0], TrilinearInterpolate(weight, &Gval[0]), TrilinearInterpolate(weight, &Gval[8]));
			unsigned char B = LinearInterpolate(weight[0], TrilinearInterpolate(weight, &Bval[0]), TrilinearInterpolate(weight, &Bval[8]));

			// Based on T-Strip coordinates, mod to edge size and insert into appropriate face
			int idx = ((i%edge) + (j%edge)*edge);
			unsigned char *ptr = imgOut[face];
			// CImg uses planar RGBA storage, hence n*edge*edge for each value
			ptr[idx + 0 * edge*edge] = R;
			ptr[idx + 1 * edge*edge] = G;
			ptr[idx + 2 * edge*edge] = B;
			ptr[idx + 3 * edge*edge] = 255;
		}
	}
}

__device__
void GetFaceStartEnd(int face, int srcWidth, int srcHeight, int edgeSize, 
	int *xStart, int *xEnd, int *yStart, int *yEnd)
{
	// back, left, front, right, top, bottom
	switch (face) {
	case 0: // back
		*xStart = 0;
		*xEnd = edgeSize;
		*yStart = edgeSize;
		*yEnd = edgeSize * 2;
		break;
	case 1: // left
		*xStart = edgeSize;
		*xEnd = edgeSize*2;
		*yStart = edgeSize;
		*yEnd = edgeSize * 2;
		break;
	case 2:
		*xStart = edgeSize * 2;
		*xEnd = edgeSize * 3;
		*yStart = edgeSize;
		*yEnd = edgeSize * 2;
		break;
	case 3:
		*xStart = edgeSize * 3;
		*xEnd = edgeSize * 4;
		*yStart = edgeSize;
		*yEnd = edgeSize * 2;
		break;
	case 4:
		*xStart = edgeSize * 2;
		*xEnd = edgeSize * 3;
		*yStart = 0;
		*yEnd = edgeSize;
		break;
	case 5:
		*xStart = edgeSize * 2;
		*xEnd = edgeSize * 3;
		*yStart = edgeSize * 2;
		*yEnd = edgeSize * 3;
		break;

	}
}


// Convert faces one at a time if we can't fit everything in memory at once
__global__
void ConvertFace(unsigned char *imgIn, unsigned char *imgOut, int face, int width, int height, int rvalue)
{
	long TstripWidth = rvalue * 4; // Use long in case we're using gigantic 32k+ images
	int edge = rvalue;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = (blockDim.x * gridDim.x);
	int xStart, xEnd, yStart, yEnd;
	GetFaceStartEnd(face, width, height, edge, &xStart, &xEnd, &yStart, &yEnd);

	int i = index + xStart;
	int maxIndex = 0;
	for (; i < xEnd; i += stride) {
		for (int j = yStart; j < yEnd; j++) {
			
			double x, y, z;
			OutImgToXYZ(i, j, face, edge, &x, &y, &z);

			double theta = atan2(y, x);
			double r = hypot(x, y);
			double phi = atan2(z, r);
			double uf = (theta + M_PI) / M_PI * height;
			double vf = (M_PI_2 - phi) / M_PI * height;
			int ui = min(static_cast<int>(std::floor(uf)), width);
			int vi = min(static_cast<int>(std::floor(vf)), height);
			int u2 = min(ui + 1, width);
			int v2 = min(vi + 1, height);
			int u3 = min(ui + 2, width);
			int v3 = min(vi + 2, height);
			int u4 = max(ui - 1, 0);
			int v4 = max(vi - 1, 0);
			int u[4] = { ui, u2, u3, u4 };
			int v[4] = { vi, v2, v3, v4 };
			double mu = uf - ui, nu = vf - vi;
			mu = nu = 0;

			unsigned char Rval[16];
			unsigned char Gval[16];
			unsigned char Bval[16];

			for (int a = 0; a < 4; a++) {
				for (int b = 0; b < 4; b++) {
					Rval[a * 4 + b] = imgIn[u[a] + v[b] * width + 0 * width*height];
					Gval[a * 4 + b] = imgIn[u[a] + v[b] * width + 1 * width*height];
					Bval[a * 4 + b] = imgIn[u[a] + v[b] * width + 2 * width*height];
				}
			}
			float weight[3] = { 0.5f, 0.5f, 0.5f };
			unsigned char R = LinearInterpolate(weight[0], TrilinearInterpolate(weight, &Rval[0]), TrilinearInterpolate(weight, &Rval[8]));
			unsigned char G = LinearInterpolate(weight[0], TrilinearInterpolate(weight, &Gval[0]), TrilinearInterpolate(weight, &Gval[8]));
			unsigned char B = LinearInterpolate(weight[0], TrilinearInterpolate(weight, &Bval[0]), TrilinearInterpolate(weight, &Bval[8]));

			// Based on T-Strip coordinates, mod to edge size and insert into appropriate face
			int idx = ((i%edge) + (j%edge)*edge);
			// CImg uses planar RGBA storage, hence n*edge*edge for each value
			imgOut[idx + 0 * edge*edge] = R;
			imgOut[idx + 1 * edge*edge] = G;
			imgOut[idx + 2 * edge*edge] = B;
			imgOut[idx + 3 * edge*edge] = 255;
		}
	}
}
#pragma endregion CUDACALLS
#endif



// Overload for nesting Interpolate calls
unsigned char Linear(float weight, unsigned char v1, unsigned char v2)
{
	return unsigned char(weight*v2 + (1.0f - weight) * v1);
}

// Expectes 1 weight and 2 values
unsigned char Linear(float weight, unsigned char *values)
{
	return unsigned char(weight * (values[1]) + (1.0f - weight)*values[0]);
}

// Expects 2 weights and 4 values
unsigned char Bilinear(float *weight, unsigned char *values)
{
	unsigned char prime[2] = {
		Linear(weight[1], &values[0]),
		Linear(weight[1], &values[2])
	};
	return Linear(weight[0], prime);
}

// Expects 3 weights and 8 values
unsigned char Trilinear(float *weight, unsigned char *values)
{
	unsigned char prime[2] = {
		Bilinear(&(weight[0]), &(values[0])),
		Bilinear(&(weight[1]), &(values[4]))
	};
	return Linear(weight[2], prime);
}

void ImgToXYZ(int i, int j, int face, int edge, double *x, double *y, double *z) {
	auto a = 2.0 * i / edge;
	auto b = 2.0 * j / edge;

	if (face == 0) { // back
		*x = -1;
		*y = 1 - a;
		*z = 3 - b;
	}
	else if (face == 1) { // left
		*x = a - 3;
		*y = -1;
		*z = 3 - b;
	}
	else if (face == 2) { // front
		*x = 1;
		*y = a - 5;
		*z = 3 - b;
	}
	else if (face == 3) { // right
		*x = 7 - a;
		*y = 1;
		*z = 3 - b;
	}
	else if (face == 4) { // top
		*x = b - 1;
		*y = a - 5;
		*z = 1;
	}
	else if (face == 5) { // bottom
		*x = 5 - b;
		*y = a - 5;
		*z = -1;
	}
}

// Convert panorama using an inverse pixel transformation on CPU
void ConvertCPU(unsigned char *imgIn, unsigned char **imgOut, int width, int height) {
	int TotalWidth = edge * 4; // Total width of the T-Strip image
	int start = 0;
	int end = 0;

	// i/j are T-Strip coordinates, that then get converted to XYZ spherical projection coordinates
#ifdef __linux__
	tbb::parallel_for(tbb::blocked_range<size_t>(0, TotalWidth, 1), [&](const tbb::blocked_range<size_t>& range) {
		for (size_t i = range.begin(); i < range.end(); i++) {
#endif
#ifdef _WIN32
	printf("TBB is not yet implemented for Windows, STL threads will be used in place of it.  This likely will not be as performant.\n");
	int maxthreads = std::thread::hardware_concurrency();
	std::thread *threads = new std::thread[maxthreads];
	for (int n = 0; n < maxthreads; n++) {
		threads[n] = std::thread([&, n](int stepping) {
			for (int i = n; i < TotalWidth; i += stepping) {
#endif
				int face = int(i / edge);
				start = (i >= 2 * edge && i < 3 * edge) ? 0 : edge;
				end = (i >= 2 * edge && i < 3 * edge) ? edge * 3 : edge * 2;

				// Range start/end determine where in the T-strip to look vertically
				for (int j = start; j < end; ++j) {
					if (j < edge) { // Check if we're above the middle of the strip, then it's the top face
						face = 4;
					}
					else if (j > 2 * edge) { // If we're below the middle of the strip, bottom face
						face = 5;
					}
					else {
						face = int(i / edge); // In the middle of the strip, determine by ratio what face we have
					}

					// Covert T-Strip coordinates to unit-cube coordinates
					double x, y, z;
					ImgToXYZ(i, j, face, edge, &x, &y, &z);

					double theta = std::atan2(y, x);
					double r = std::hypot(x, y);
					double phi = std::atan2(z, r);
					double uf = (theta + M_PI) / M_PI * height;
					double vf = (M_PI_2 - phi) / M_PI * height;
					/*
						Coordinate structure:
						[     ][     ][     ][u3/v4]
						[     ][     ][u2/v2][     ]
						[     ][ui/vi][     ][     ]
						[u4/v4][     ][     ][     ]
					*/
					// Width/Height - 1 max to prevent OOB errors
					int ui = std::min(static_cast<int>(std::floor(uf)), width-1);
					int vi = std::min(static_cast<int>(std::floor(vf)), height-1);
					int u2 = std::min(ui + 1, width-1);
					int v2 = std::min(vi + 1, height-1);
					int u3 = std::min(ui + 2, width-1);
					int v3 = std::min(vi + 2, height-1);
					int u4 = std::max(ui - 1, 0);
					int v4 = std::max(vi - 1, 0);
					int u[4] = { ui, u2, u3, u4 };
					int v[4] = { vi, v2, v3, v4 };

					unsigned char Rval[16];
					unsigned char Gval[16];
					unsigned char Bval[16];

					for (int a = 0; a < 4; a++) {
						for (int b = 0; b < 4; b++) {
							Rval[a * 4 + b] = imgIn[u[a] + v[b] * width + 0 * width*height];
							Gval[a * 4 + b] = imgIn[u[a] + v[b] * width + 1 * width*height];
							Bval[a * 4 + b] = imgIn[u[a] + v[b] * width + 2 * width*height];
						}
					}

					float weight[3] = { 0.5f, 0.5f, 0.5f };
					unsigned char R = Linear(weight[0], Trilinear(weight, &Rval[0]), Trilinear(weight, &Rval[8]));
					unsigned char G = Linear(weight[0], Trilinear(weight, &Gval[0]), Trilinear(weight, &Gval[8]));
					unsigned char B = Linear(weight[0], Trilinear(weight, &Bval[0]), Trilinear(weight, &Bval[8]));

					// Based on T-Strip coordinates, mod to edge size and insert into appropriate face
					int idx = ((i%edge) + (j%edge)*edge);
					// CImg uses planar RGBA storage, hence n*edge*edge for each value
					imgOut[face][idx + 0 * edge*edge] = R;
					imgOut[face][idx + 1 * edge*edge] = G;
					imgOut[face][idx + 2 * edge*edge] = B;
					imgOut[face][idx + 3 * edge*edge] = 255;
				}
			}
#ifdef _WIN32
		}, maxthreads);
#endif
	}
#ifdef __linux__
	});
#endif
	// macros, macros everywhere
#ifdef _WIN32
	for (int i = 0; i < maxthreads; i++) {
		threads[i].join();
	}
	delete[]threads;
#endif
}

// Transform equirectangular panorama to new one
void TransformCPU(unsigned char *imgIn, unsigned char *imgOut, int width, int height, double yaw, double pitch, double roll)
{
	int threadCount = std::thread::hardware_concurrency();
	std::thread *threads = new std::thread[threadCount];
	for (int n = 0; n < threadCount; n++) {
		threads[n] = std::thread([&, n]() {
			for (int x = n; x < width; x += threadCount) {
				for (int y = 0; y < height; y++) {
					double xx = 2.0 * (x + 0.5) / width - 1.0;
					double yy = 2.0 * (y + 0.5) / height - 1.0;
					double lng = M_PI * xx;
					double lat = M_PI_2 * yy;
					double X, Y, Z, D;
					int ix, iy;

					X = cos(lat) * cos(lng);
					Y = cos(lat) * sin(lng);
					Z = sin(lat);
					D = sqrt(X * X + Y * Y);

					glm::mat4 rotation = glm::rotate(glm::mat4(1), glm::radians((float)yaw), glm::vec3(0.0, 0.0, 1.0))
						* glm::rotate(glm::mat4(1), glm::radians((float)pitch), glm::vec3(0.0, 1.0, 0.0))
						* glm::rotate(glm::mat4(1), glm::radians((float)roll), glm::vec3(1.0, 0.0, 0.0));
					glm::vec3 outXYZ = glm::vec3(rotation * glm::vec4(X, Y, Z, 1.0));
					X = outXYZ.x;
					Y = outXYZ.y;
					Z = outXYZ.z;

					lat = atan2(Z, D);
					lng = atan2(Y, X);

					ix = (0.5 * lng / M_PI + 0.5) * width - 0.5;
					iy = (lat / M_PI + 0.5) * height - 0.5;

					int ui = std::min(ix, width - 1);
					int vi = std::min(iy, height - 1);
					int u2 = std::min(ui + 1, width - 1);
					int v2 = std::min(vi + 1, height - 1);
					int u3 = std::min(ui + 2, width - 1);
					int v3 = std::min(vi + 2, height - 1);
					int u4 = std::max(ui - 1, 0);
					int v4 = std::max(vi - 1, 0);
					int u[4] = { ui, u2, u3, u4 };
					int v[4] = { vi, v2, v3, v4 };

					unsigned char Rval[16];
					unsigned char Gval[16];
					unsigned char Bval[16];
					for (int a = 0; a < 4; a++) {
						for (int b = 0; b < 4; b++) {
							Rval[a * 4 + b] = imgIn[u[a] + v[b] * width + 0 * width*height];
							Gval[a * 4 + b] = imgIn[u[a] + v[b] * width + 1 * width*height];
							Bval[a * 4 + b] = imgIn[u[a] + v[b] * width + 2 * width*height];
						}
					}

					float weight[3] = { 0.5f, 0.5f, 0.5f };
					unsigned char R = Linear(weight[0], Trilinear(weight, &Rval[0]), Trilinear(weight, &Rval[8]));
					unsigned char G = Linear(weight[0], Trilinear(weight, &Gval[0]), Trilinear(weight, &Gval[8]));
					unsigned char B = Linear(weight[0], Trilinear(weight, &Bval[0]), Trilinear(weight, &Bval[8]));
					imgOut[x + y * width + 0 * width*height] = R;
					imgOut[x + y * width + 1 * width*height] = G;
					imgOut[x + y * width + 2 * width*height] = B;
				}
			}
		});
	}
	for (int n = 0; n < threadCount; n++) {
		threads[n].join();
	}
	delete[]threads;
}

int parseParameters(int argc, char *argv[]) {
	iflag = oflag = hflag = rflag = cflag = 0;
	ivalue = ovalue = NULL;
	int c;

#ifdef _WIN32
	for (int i = 1; i < argc; i++) {
		if (argv[i] == std::string("-i")) {
			iflag = 1;
			ivalue = argv[++i];
		}
		if (argv[i] == std::string("-o")) {
			oflag = 1;
			ovalue = argv[++i];
		}
		if (argv[i] == std::string("-r")) {
			rflag = 1;
			edge = std::stoi(argv[++i]);
		}
		if (argv[i] == std::string("-c")) {
			cflag = 1;
		}
		if (argv[i] == std::string("-t")) {
			tflag = 1;
			yaw = std::stod(argv[++i]);
			pitch = std::stod(argv[++i]);
			roll = std::stod(argv[++i]);
		}
		if (argv[i] == std::string("-h")) {
			fprintf(stderr, "Usage:\n\t -i <input file>\n\t -o <output file(s)>\n\t -r <edge size>\n-t <yaw> <pitch> <roll>\n\t -c (enable CUDA)\n");
			abort();
		}
	}
#elif __linux__
	opterr = 0;
	while ((c = getopt(argc, argv, "i:o:r:")) != -1)
		switch (c) {
		case 'i':
			// input file
			iflag = 1;
			ivalue = optarg;
			break;
		case 'o':
			oflag = 1;
			ovalue = optarg;
			break;
		case 'r':
			rflag = 1;
			edge = std::stoi(optarg);
			break;
		case 'c':
			printf("-c flag detected.  CUDA is not yet implemented for Linux systems, processing will take place on CPU instead\n");
			cflag = 1;
			break;
		case '?':
			if (optopt == 'i' || optopt == 'o' || optopt == 'r')
				fprintf(stderr, "Option -%c requires an argument.\n", optopt);
			else if (isprint(optopt))
				fprintf(stderr, "Unknown option `-%c'.\n", optopt);
			else
				fprintf(stderr, "Unknown option character `\\x%x'.\n", optopt);
			return 1;
		default:
			abort();
		}
#endif

	if (iflag == 0 || oflag == 0) {
		fprintf(stderr, "No inputs or outputs specified: %d / %d\n", iflag, oflag);
		abort();
		return 1;
	}
	return 0;
}


int main(int argc, char *argv[])
{
	parseParameters(argc, argv);
	printf("Converting [%s] to faces [%s] with size [%d]...\n", ivalue, ovalue, edge);

	// Load input Image
	CImg<unsigned char> ImgIn(ivalue);
	printf("%d\n", ImgIn.size());

	if (tflag) {
		CImg<unsigned char> CImgOut(ImgIn.width(), ImgIn.height(), 1, 3, 255);
		TransformCPU(ImgIn.data(), CImgOut.data(), ImgIn.width(), ImgIn.height(), yaw, pitch, roll);
		CImgOut.save_png("Out.png");
		return 0;
	}

	// Create output Images
	CImg<unsigned char>* CImgOut[6];
	unsigned char* imgOut[6];
	for (int i = 0; i < 6; ++i) {
		CImgOut[i] = new CImg<unsigned char>(edge, edge, 1, 4, 255);
		imgOut[i] = (unsigned char*)CImgOut[i]->data();
	}

	// Start timer
	std::chrono::high_resolution_clock::time_point total = std::chrono::high_resolution_clock::now();
#ifdef _WIN32
	if (cflag) {
		printf("Using CUDA for processing\n");

		// Store these values on the stack
		int InSize = ImgIn.size();
		int width = ImgIn.width();
		int height = ImgIn.height();

		// Allocate space on the device for the equirectangular image
		unsigned char *d_ImgIn;
		HANDLE_ERROR(hipMallocManaged((void**)&d_ImgIn, InSize * sizeof(unsigned char)));
		std::memcpy(d_ImgIn, ImgIn.data(), InSize * sizeof(unsigned char));

		// Clear the input image from host memory, we never use it CPU side again
		ImgIn.clear();

		// Check memory availability
		size_t total_memory, free_memory;
		HANDLE_ERROR(hipMemGetInfo(&free_memory, &total_memory) );
		printf("Total Memory: %lld\n Free Memory: %lld\n", total_memory, free_memory);

		long outSize = edge * edge * 4;
		printf("Insize memory: %d\n Outsize memory each: %lld\n Outsize memory total: %lld\n", InSize, outSize, outSize * 6);

		if (outSize > free_memory) {
			fprintf(stderr, "Not enough memory free on GPU device, please run without -c flag to compute on CPU");
			return -1;
		}

		// See if we can fit all 6 cubeface output images into device memory at once, if not render them sequentially
		if (outSize * 6 > free_memory) {
			std::thread threads[6];
			fprintf(stderr, "Not enough memory on device to run all faces in parallel, running one face at a time\n");
			unsigned char *d_ImgOut, *h_ImgOut;

			// Malloc a single ImgOut on the device to be reused for each face
			HANDLE_ERROR(hipMalloc((void**)&d_ImgOut, outSize * sizeof(unsigned char)));
			// hipHostMalloc allocates non-paging CPU-side memory and greatly speeds up memcpy operations
			HANDLE_ERROR(hipHostMalloc((void**)&h_ImgOut, outSize * sizeof(unsigned char)));
			std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

			// Block sizes should be in multiples of warps (32)
			int blockSize = 256;
			// Calculate number of blocks to give us maximal parallelization
			int blockCount = (edge + blockSize - 1) / blockSize;
			printf("Total threads: %d\n", blockSize * blockCount);

			for (int i = 0; i < 6; i++) {
				fprintf(stderr, "Starting face %d...\n", i);
				ConvertFace <<<blockCount, blockSize >>> (d_ImgIn, d_ImgOut, i, width, height, edge);
				HANDLE_ERROR(hipDeviceSynchronize());
				HANDLE_ERROR(hipDeviceSynchronize());
				HANDLE_ERROR(hipMemcpy(h_ImgOut, d_ImgOut, outSize * sizeof(unsigned char), hipMemcpyDeviceToHost));
				std::memcpy(imgOut[i], h_ImgOut, outSize * sizeof(unsigned char));

				// Image data now lives CPU-side, immediately start a thread to begin writing it to disk while we process the others
				threads[i] = std::thread([&, i]() {
					std::string fname = std::string(ovalue) + "_" + std::to_string(i) + ".png";
					CImgOut[i]->save_png(fname.c_str());
					CImgOut[i]->clear();
					printf("Thread %d finished writing to disk\n", i);
				});
			}
			// Free everything we don't use anymore
			HANDLE_ERROR(hipFree(d_ImgIn));
			HANDLE_ERROR(hipFree(d_ImgOut));
			HANDLE_ERROR(hipHostFree(h_ImgOut));

			fprintf(stderr, "Time to convert: %lldms\n", std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - t1).count());
			for (int i = 0; i < 6; i++) {
				threads[i].join();
				printf("Joined thread %d\n", i);
			}
			fprintf(stderr, "Total Time To Convert And Write: %lldms\n", std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - total).count());
		}
		else {
			printf("Allocating GPU memory\n");
			unsigned char **d_Faces;
			HANDLE_ERROR(hipMallocManaged((void**)&d_Faces, sizeof(unsigned char*) * 6));
			for (int i = 0; i < 6; i++) {
				HANDLE_ERROR(hipMallocManaged((void**)&d_Faces[i], outSize * sizeof(unsigned char)));
				HANDLE_ERROR(hipMemGetInfo(&free_memory, &total_memory));
				printf("Free Device Memory: %lldMB\n", free_memory / 1024 / 1024);
			}

			std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

			int blockSize = 256;
			int blockCount = (width + blockSize - 1) / blockSize;
			fprintf(stderr, "Starting conversion...\n");

			ConvertBack <<<blockCount, blockSize >>> (d_ImgIn, d_Faces, width, height, edge);
			HANDLE_ERROR(hipDeviceSynchronize());
			HANDLE_ERROR(hipDeviceSynchronize());
			HANDLE_ERROR(hipFree(d_ImgIn));

			fprintf(stderr, "Time to convert: %lldms\n", std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - t1).count());
			fprintf(stderr, "Device Synchronized\n");


			std::thread threads[6];
			for (int i = 0; i < 6; i++) {
				// Copy image data to host memory, then free it from the device
				std::memcpy(imgOut[i], d_Faces[i], outSize * sizeof(unsigned char));
				HANDLE_ERROR(hipFree(d_Faces[i]));
				threads[i] = std::thread([&, i]() {
					std::string fname = std::string(ovalue) + "_" + std::to_string(i) + ".png";
					printf("Writing %s to disk\n", fname.c_str());
					CImgOut[i]->save_png(fname.c_str());
					CImgOut[i]->clear();
					printf("Finished writing %s\n", fname.c_str());
				});
			}
			HANDLE_ERROR(hipFree(d_Faces));
			for (int i = 0; i < 6; i++) {
				threads[i].join();
			}

			fprintf(stderr, "Total Time To Convert And Write: %lldms\n", std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - total).count());
		}
	}
	else {
#endif
		printf("Using CPU for Processing\n");
		// Note: With TBB it's faster to process the entire T-Strip at once because we may have more than 6 available threads
		// If we do, then doing each face iteratively and threading the save routine (the slow part) will end up being much slower than threading all 6 save calls at once
		ConvertCPU(ImgIn.data(), imgOut, ImgIn.width(), ImgIn.height());
		std::thread threads[6];
		for (int i = 0; i < 6; i++) {
			threads[i] = std::thread([&, i]() {
				std::string fname = std::string(ovalue) + "_" + std::to_string(i) + ".png";
				printf("Writing %s to disk\n", fname.c_str());
				CImgOut[i]->save_png(fname.c_str());
				CImgOut[i]->clear();
				printf("Finished writing %s\n", fname.c_str());
			});
		}
		for (int i = 0; i < 6; i++) {
			threads[i].join();
		}
		fprintf(stderr, "Total Time To Convert And Write: %lldms\n", std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - total).count());
#ifdef _WIN32
	}
#endif

	return 0;
}



